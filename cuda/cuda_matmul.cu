#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include "../sparse_matrix.h"
#include "cuda_matmul.h"


#define DEBUG
#ifdef DEBUG
#define cudaCheckError(ans) cudaAssert((ans), __FILE__, __LINE__);
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
if (code != hipSuccess)
{
fprintf(stderr, "CUDA Error: %s at %s:%d\n",
hipGetErrorString(code), file, line);
if (abort) exit(code);
}
}
#else
#define cudaCheckError(ans) ans
#endif

__global__ void csrMatMulKernel_naive(int rowsA, int colsB, 
                                      int *A_row_ptr, int *A_col_idx, double *A_values, 
                                      int *B_row_ptr, int *B_col_idx, double *B_values, 
                                      int *C_row_ptr, int *C_col_idx, double *C_values) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rowsA) {
        double value;

        // i indicates the index of the non-zero value in A we currently are
        for (int i = A_row_ptr[row]; i < A_row_ptr[row + 1]; i++) {
            // A_col indicates the corresponding row in B
            int A_col = A_col_idx[i];  
            double A_val = A_values[i]; 

            // j indicates the index of the non_zero value in B we currently are
            for (int j = B_row_ptr[A_col]; j < B_row_ptr[A_col + 1]; j++) {
                // B_col gives the column index of the current non-zero value in B
                int B_col = B_col_idx[j];   
                double B_val = B_values[j]; 

                // Multiply and accumulate into C
                value = A_val * B_val;
                if (value != 0){
                    C_col_idx[row * colsB + B_col] = B_col;
                    C_row_ptr[row + 1] += 1;
                    C_values[row * colsB + B_col] = value;
                }
            }
        }
    }
}

__global__ void cooMatMulKernel_naive(int nnzA, int *rowA, int *colA, double *valA,
                                int nnzB, int *rowB, int *colB, double *valB,
                                int *rowC, int *colC, double *valC, int *nnzC) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= nnzA) return;

    int row = rowA[idx];
    int col = colA[idx];
    double val = valA[idx];

    // Iterate over all non-zero elements in matrix B
    for (int j = 0; j < nnzB; j++) {
        if (rowB[j] == col) { // Matching row in B with column in A
            int resultRow = row;
            int resultCol = colB[j];
            double resultVal = val * valB[j];

            // Atomic addition to ensure correctness for parallel writes
            int resultIdx = atomicAdd(nnzC, 1);
            rowC[resultIdx] = resultRow;
            colC[resultIdx] = resultCol;
            valC[resultIdx] = resultVal;
        }
    }
}



extern "C" void multiplyCSR_CUDA(const CSRMatrix &A, CSRMatrix &B, CSRMatrix &C) {

    int threadsPerBlock = 512;
    int blocksPerGrid = (C.rows + threadsPerBlock - 1) / threadsPerBlock;

    // Allocate device memory
    int *d_B_row_ptr, *d_B_col_idx;
    double *d_B_values;

    int *d_A_row_ptr, *d_A_col_idx, *d_C_row_ptr;
    double *d_A_values, *d_C_values;
    int *d_C_col_idx;

    // Allocate and copy CSR data for A
    hipMalloc(&d_A_row_ptr, A.row_ptr.size() * sizeof(int));
    hipMalloc(&d_A_col_idx, A.col_idx.size() * sizeof(int));
    hipMalloc(&d_A_values, A.values.size() * sizeof(double));
    hipMemcpy(d_A_row_ptr, A.row_ptr.data(), A.row_ptr.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_A_col_idx, A.col_idx.data(), A.col_idx.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_A_values, A.values.data(), A.values.size() * sizeof(double), hipMemcpyHostToDevice);

    // Allocate and copy CSR data for B
    hipMalloc(&d_B_row_ptr, B.row_ptr.size() * sizeof(int));
    hipMalloc(&d_B_col_idx, B.col_idx.size() * sizeof(int));
    hipMalloc(&d_B_values, B.values.size() * sizeof(double));
    hipMemcpy(d_B_row_ptr, B.row_ptr.data(), B.row_ptr.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B_col_idx, B.col_idx.data(), B.col_idx.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B_values, B.values.data(), B.values.size() * sizeof(double), hipMemcpyHostToDevice);

    // Allocate memory for C
    hipMalloc(&d_C_row_ptr, (C.rows + 1) * sizeof(int));
    hipMalloc(&d_C_col_idx, (A.rows * B.cols) * sizeof(int));  // Estimate non-zero count for C
    hipMalloc(&d_C_values, (A.rows * B.cols) * sizeof(double));

    // Kernel launch
    csrMatMulKernel_naive<<<blocksPerGrid, threadsPerBlock>>>(
        A.rows, B.cols, d_A_row_ptr, d_A_col_idx, d_A_values, 
        d_B_row_ptr, d_B_col_idx, d_B_values, 
        d_C_row_ptr, d_C_col_idx, d_C_values);

    // Copy results back to host
    hipMemcpy(C.row_ptr.data(), d_C_row_ptr, (C.rows + 1) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(C.col_idx.data(), d_C_col_idx, (A.rows * B.cols) * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(C.values.data(), d_C_values, (A.rows * B.cols) * sizeof(double), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A_row_ptr);
    hipFree(d_A_col_idx);
    hipFree(d_A_values);
    hipFree(d_B_row_ptr);
    hipFree(d_B_col_idx);
    hipFree(d_B_values);
    hipFree(d_C_row_ptr);
    hipFree(d_C_col_idx);
    hipFree(d_C_values);
}


extern "C" void multiplyCOO_CUDA(const COOMatrix &A, const COOMatrix &B, COOMatrix &C) {
    int *d_rowA, *d_colA, *d_rowB, *d_colB, *d_rowC, *d_colC;
    double *d_valA, *d_valB, *d_valC;
    int *d_nnzC;

    int nnzC_host = 0;  // Initial non-zero count for matrix C

    // Allocate device memory for A
    hipMalloc(&d_rowA, A.row_idx.size() * sizeof(int));
    hipMalloc(&d_colA, A.col_idx.size() * sizeof(int));
    hipMalloc(&d_valA, A.values.size() * sizeof(double));
    hipMemcpy(d_rowA, A.row_idx.data(), A.row_idx.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_colA, A.col_idx.data(), A.col_idx.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_valA, A.values.data(), A.values.size() * sizeof(double), hipMemcpyHostToDevice);

    // Allocate device memory for B
    hipMalloc(&d_rowB, B.row_idx.size() * sizeof(int));
    hipMalloc(&d_colB, B.col_idx.size() * sizeof(int));
    hipMalloc(&d_valB, B.values.size() * sizeof(double));
    hipMemcpy(d_rowB, B.row_idx.data(), B.row_idx.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_colB, B.col_idx.data(), B.col_idx.size() * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_valB, B.values.data(), B.values.size() * sizeof(double), hipMemcpyHostToDevice);

    // Allocate device memory for C (result)
    hipMalloc(&d_rowC, A.row_idx.size() * B.col_idx.size() * sizeof(int));  // Overestimate
    hipMalloc(&d_colC, A.row_idx.size() * B.col_idx.size() * sizeof(int));
    hipMalloc(&d_valC, A.row_idx.size() * B.col_idx.size() * sizeof(double));
    hipMalloc(&d_nnzC, sizeof(int));
    hipMemcpy(d_nnzC, &nnzC_host, sizeof(int), hipMemcpyHostToDevice);

    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (A.row_idx.size() + threadsPerBlock - 1) / threadsPerBlock;
    cooMatMulKernel_naive <<<blocksPerGrid, threadsPerBlock>>>(
        A.row_idx.size(), d_rowA, d_colA, d_valA,
        B.row_idx.size(), d_rowB, d_colB, d_valB,
        d_rowC, d_colC, d_valC, d_nnzC);

    // Copy back results
    hipMemcpy(&nnzC_host, d_nnzC, sizeof(int), hipMemcpyDeviceToHost);
    C.row_idx.resize(nnzC_host);
    C.col_idx.resize(nnzC_host);
    C.values.resize(nnzC_host);

    hipMemcpy(C.row_idx.data(), d_rowC, nnzC_host * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(C.col_idx.data(), d_colC, nnzC_host * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(C.values.data(), d_valC, nnzC_host * sizeof(double), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_rowA);
    hipFree(d_colA);
    hipFree(d_valA);
    hipFree(d_rowB);
    hipFree(d_colB);
    hipFree(d_valB);
    hipFree(d_rowC);
    hipFree(d_colC);
    hipFree(d_valC);
    hipFree(d_nnzC);
}
